#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#ifndef BLOCK_DIM_Y
#define BLOCK_DIM_Y 1024 
#endif

#ifndef UNROLL_FACTOR
#define UNROLL_FACTOR 8
#endif
constexpr int URF{UNROLL_FACTOR};

#ifndef SOFTMAX_VARIANT
#define SOFTMAX_VARIANT 8
#endif

#ifndef WIDTH
#define WIDTH 0
#endif

#define CEILING(x,y) (((x) + (y) - 1) / (y))
/*
template <typename scalar_t>
__global__ void softmax_kernel(scalar_t* __restrict__ a, scalar_t* __restrict__ b, int w, int h)
{
  int col = blockIdx.x*blockDim.x + threadIdx.x;
  int row = blockIdx.y*blockDim.y + threadIdx.y;
  if (row < h && col < w)
  {
    float maxval = a[row*w];
    for (int i = 1; i<w; i++)
    {
      maxval = fmaxf(maxval, a[row*w + i]);
    }
    float divisor = 0.f;
    for (int i = 0; i<w; i++)
    {
      divisor += __expf(a[row*w + i] - maxval);
    }
    b[row*w + col] = __expf(a[row*w + col]-maxval)/(divisor);
  }
}


template <typename scalar_t>
__global__ void softmax_kernel2(scalar_t* __restrict__ a, scalar_t* __restrict__ b, int w, int h)
{
  int row = blockIdx.x*blockDim.x + threadIdx.x;
  int ty = threadIdx.y;
  __shared__ float reduction[BLOCK_DIM_Y]; 
  if (row < h)
  {
    float maxval = 0;
    for (int i = ty*BLOCK_DIM_Y; i<min(w, (ty+1)*BLOCK_DIM_Y); i+=1)
    {
      maxval = fmaxf(maxval, a[row*w + i]);
    }

    reduction[ty] = maxval;
    for(int stride = BLOCK_DIM_Y/2; stride>=1; stride/=2)
    {
      __syncthreads();
      if (ty < stride)
      {
        reduction[ty] = fmaxf(reduction[ty], reduction[ty+stride]);
      }
    }

    __syncthreads();
    maxval = reduction[0];
    float divisor = 0.f;
    for (int i = ty*BLOCK_DIM_Y; i<min(w, (ty+1)*BLOCK_DIM_Y); i+=1)
    {
      divisor += __expf(a[row*w + i] - maxval);
    }
    reduction[ty] = divisor;
    for(int stride = BLOCK_DIM_Y/2; stride>=1; stride/=2)
    {
      __syncthreads();
      if (ty < stride)
      {
        reduction[ty] = reduction[ty] + reduction[ty+stride];
      }
    }
    __syncthreads();
    divisor = reduction[0];

    for (int i = ty; i<w; i+=BLOCK_DIM_Y)
    {
      b[row*w + i] = __expf(a[row*w + i]-maxval)/divisor;
    }

  }
}

template <typename scalar_t>
__global__ void softmax_kernel3(scalar_t* __restrict__ a, scalar_t* __restrict__ b, int w, int h)
{
  int row = blockIdx.x*blockDim.x + threadIdx.x;
  int ty = threadIdx.y;
  __shared__ float reduction[BLOCK_DIM_Y]; 
  if (row < h)
  {
    float maxval = 0;
    for (int i = ty; i<w; i+=BLOCK_DIM_Y)
    {
      maxval = fmaxf(maxval, a[row*w + i]);
    }

    reduction[ty] = maxval;
    for(int stride = BLOCK_DIM_Y/2; stride>=1; stride/=2)
    {
      __syncthreads();
      if (ty < stride)
      {
        reduction[ty] = fmaxf(reduction[ty], reduction[ty+stride]);
      }
    }

    __syncthreads();
    maxval = reduction[0];

    float divisor = 0.f;
    for (int i = ty; i<w; i+=BLOCK_DIM_Y)
    {
      divisor += __expf(a[row*w + i] - maxval);
    }
    reduction[ty] = divisor;
    for(int stride = BLOCK_DIM_Y/2; stride>=1; stride/=2)
    {
      __syncthreads();
      if (ty < stride)
      {
        reduction[ty] = reduction[ty] + reduction[ty+stride];
      }
    }
    __syncthreads();
    divisor = reduction[0];

    for (int i = ty; i<w; i+=BLOCK_DIM_Y)
    {
      b[row*w + i] = __expf(a[row*w + i]-maxval)/divisor;
    }
  }
}
*/
template <typename scalar_t>
__global__ void softmax_kernel4(scalar_t* __restrict__ a, scalar_t* __restrict__ b, int w, int h)
{
  int row = blockIdx.x*blockDim.x + threadIdx.x;
  int ty = threadIdx.y;
  int warp_id = ty/32;
  __shared__ float reduction[BLOCK_DIM_Y/32]; 
  if (row < h)
  {
    float maxval = 0;
    for (int i = ty; i<w; i+=BLOCK_DIM_Y)
    {
      maxval = fmaxf(maxval, a[row*w + i]);
    }
    for (int mask = 16; mask>0; mask/=2)
    {
      maxval = fmaxf(maxval, __shfl_xor_sync(0xffffffff, maxval, mask, 32));
    }

    if (ty%32 == 0)
    {
      reduction[warp_id] = maxval;
    }
    __syncthreads();
    if (warp_id == 0)
    {
        maxval = ty < BLOCK_DIM_Y/32 ? reduction[ty] : 0;
        for (int mask = 16; mask>0; mask/=2)
        {
          maxval = fmaxf(maxval, __shfl_xor_sync(0xffffffff, maxval, mask, 32));
        }
    }
    if (ty == 0)
    {
        reduction[0] = maxval;
    }
    __syncthreads();
    maxval = reduction[0];
    float divisor = 0.f;
    for (int i = ty; i<w; i+=BLOCK_DIM_Y)
    {
      divisor += __expf(a[row*w + i] - maxval);
    }
    for (int mask = 16; mask>0; mask/=2)
    {
      divisor += __shfl_xor_sync(0xffffffff, divisor, mask, 32);
    }

    if (ty%32 == 0)
    {
      reduction[warp_id] = divisor;
    }

    __syncthreads();
    if (warp_id == 0)
    {
        divisor = ty < BLOCK_DIM_Y/32 ? reduction[ty] : 0;
        for (int mask = 16; mask>0; mask/=2)
        {
          divisor += __shfl_xor_sync(0xffffffff, divisor, mask, 32);
        }
    }
    if (ty == 0)
    {
        reduction[0] = divisor;
    }

    __syncthreads();
    divisor = reduction[0];

    for (int i = ty; i<w; i+=BLOCK_DIM_Y)
    {
      b[row*w + i] = __expf(a[row*w + i]-maxval)/divisor;
    }
  }
}
/*
template <typename scalar_t>
__global__ void softmax_kernel5(scalar_t* __restrict__ a, scalar_t* __restrict__ b, int w, int h)
{
  int row = blockIdx.x*blockDim.x + threadIdx.x;
  int ty = threadIdx.y;
  __shared__ float reduction[BLOCK_DIM_Y/2]; 
  if (row < h)
  {
    float maxval = 0;
    for (int i = ty; i<w/4; i+=BLOCK_DIM_Y)
    {
      float4 val = reinterpret_cast<float4*>(&a[row*w + i*4])[0];
      maxval = fmaxf(maxval, val.x);
      maxval = fmaxf(maxval, val.y);
      maxval = fmaxf(maxval, val.z);
      maxval = fmaxf(maxval, val.w);
    }

    if (ty >= BLOCK_DIM_Y/2)
    {
      reduction[ty - BLOCK_DIM_Y/2] = maxval;
    }
    for(int stride = BLOCK_DIM_Y/2; stride>=1; stride/=2)
    {
      __syncthreads();
      if (ty < stride)
      {
        maxval = fmaxf(maxval, reduction[ty]);
        if (ty >= stride/2)
        {
          reduction[ty - stride/2] = maxval;
        }
      }
    }

    __syncthreads();
    maxval = reduction[0];

    float divisor = 0.f;
    for (int i = ty; i<w/4; i+=BLOCK_DIM_Y)
    {
      float4 val = reinterpret_cast<float4*>(&a[row*w + i*4])[0];
      divisor += __expf(val.x - maxval);
      divisor += __expf(val.y - maxval);
      divisor += __expf(val.z - maxval);
      divisor += __expf(val.w - maxval);
    }

    if (ty >= BLOCK_DIM_Y/2)
    {
      reduction[ty - BLOCK_DIM_Y/2] = divisor;
    }

    for(int stride = BLOCK_DIM_Y/2; stride>=1; stride/=2)
    {
      __syncthreads();
      if (ty < stride)
      {
        divisor = divisor + reduction[ty];
        if (ty >= stride/2)
        {
          reduction[ty - stride/2] = divisor;
        }
      }
    }
    __syncthreads();
    divisor = reduction[0];

    for (int i = ty; i<w/4; i+=BLOCK_DIM_Y)
    {
        float4 val = reinterpret_cast<float4*>(&a[row*w + i*4])[0];
        val.x = __expf(val.x-maxval)/divisor;
        val.y = __expf(val.y-maxval)/divisor;
        val.z = __expf(val.z-maxval)/divisor;
        val.w = __expf(val.w-maxval)/divisor;
        reinterpret_cast<float4*>(&b[row*w + i*4])[0] = val;
    }
  }
}

template <typename scalar_t>
__global__ void softmax_kernel6(scalar_t* __restrict__ a, scalar_t* __restrict__ b, int w, int h)
{
  int row = blockIdx.x*blockDim.x + threadIdx.x;
  int ty = threadIdx.y;
  int warp_id = ty/32;
  __shared__ float reduction[BLOCK_DIM_Y/32]; 
  if (row < h)
  {
    float maxval = 0;
    for (int i = ty; i<w/4; i+=BLOCK_DIM_Y)
    {
      float4 val = reinterpret_cast<float4*>(&a[row*w + i*4])[0];
      maxval = fmaxf(maxval, val.x);
      maxval = fmaxf(maxval, val.y);
      maxval = fmaxf(maxval, val.z);
      maxval = fmaxf(maxval, val.w);
    }
    for (int mask = 16; mask>0; mask/=2)
    {
      maxval = fmaxf(maxval, __shfl_xor_sync(0xffffffff, maxval, mask, 32));
    }

    if (ty%32 == 0)
    {
      reduction[warp_id] = maxval;
    }
    __syncthreads();
    if (warp_id == 0)
    {
        maxval = ty < BLOCK_DIM_Y/32 ? reduction[ty] : 0;
        for (int mask = 16; mask>0; mask/=2)
        {
          maxval = fmaxf(maxval, __shfl_xor_sync(0xffffffff, maxval, mask, 32));
        }
    }
    if (ty == 0)
    {
        reduction[0] = maxval;
    }
    __syncthreads();
    maxval = reduction[0];
    float divisor = 0.f;
    for (int i = ty; i<w/4; i+=BLOCK_DIM_Y)
    {
      float4 val = reinterpret_cast<float4*>(&a[row*w + i*4])[0];
      divisor += __expf(val.x - maxval);
      divisor += __expf(val.y - maxval);
      divisor += __expf(val.z - maxval);
      divisor += __expf(val.w - maxval);
    }
    for (int mask = 16; mask>0; mask/=2)
    {
      divisor += __shfl_xor_sync(0xffffffff, divisor, mask, 32);
    }

    if (ty%32 == 0)
    {
      reduction[warp_id] = divisor;
    }

    __syncthreads();
    if (warp_id == 0)
    {
        divisor = ty < BLOCK_DIM_Y/32 ? reduction[ty] : 0;
        for (int mask = 16; mask>0; mask/=2)
        {
          divisor += __shfl_xor_sync(0xffffffff, divisor, mask, 32);
        }
    }
    if (ty == 0)
    {
        reduction[0] = divisor;
    }

    __syncthreads();
    divisor = reduction[0];

    for (int i = ty; i<w/4; i+=BLOCK_DIM_Y)
    {
        float4 val = reinterpret_cast<float4*>(&a[row*w + i*4])[0];
        val.x = __expf(val.x-maxval)/divisor;
        val.y = __expf(val.y-maxval)/divisor;
        val.z = __expf(val.z-maxval)/divisor;
        val.w = __expf(val.w-maxval)/divisor;
        reinterpret_cast<float4*>(&b[row*w + i*4])[0] = val;
    }
  }
}

    template <typename scalar_t>
__global__ void softmax_kernel7(scalar_t* __restrict__ a, scalar_t* __restrict__ b, int w, int h)
{
  int row = blockIdx.x;
  int ty = threadIdx.y;
  int warp_id = ty/32;
  __shared__ float reduction[BLOCK_DIM_Y/32]; 
  if (row < h)
  {
    float maxval = 0;
#pragma unroll URF
    for (int i = ty; i<w/4; i+=BLOCK_DIM_Y)
    {
        float4 val = reinterpret_cast<float4*>(&a[row*w + i*4])[0];
        maxval = fmaxf(maxval, val.x);
        maxval = fmaxf(maxval, val.y);
        maxval = fmaxf(maxval, val.z);
        maxval = fmaxf(maxval, val.w);
    }
    maxval = fmaxf(maxval, __shfl_down_sync(0xffffffff, maxval, 16, 32));
    maxval = fmaxf(maxval, __shfl_down_sync(0xffffffff, maxval, 8, 32));
    maxval = fmaxf(maxval, __shfl_down_sync(0xffffffff, maxval, 4, 32));
    maxval = fmaxf(maxval, __shfl_down_sync(0xffffffff, maxval, 2, 32));
    maxval = fmaxf(maxval, __shfl_down_sync(0xffffffff, maxval, 1, 32));

    if (ty%32 == 0)
    {
      reduction[warp_id] = maxval;
    }
    __syncthreads();
    if (warp_id == 0)
    {
        maxval = ty < BLOCK_DIM_Y/32 ? reduction[ty] : 0;
        maxval = fmaxf(maxval, __shfl_down_sync(0xffffffff, maxval, 16, 32));
        maxval = fmaxf(maxval, __shfl_down_sync(0xffffffff, maxval, 8, 32));
        maxval = fmaxf(maxval, __shfl_down_sync(0xffffffff, maxval, 4, 32));
        maxval = fmaxf(maxval, __shfl_down_sync(0xffffffff, maxval, 2, 32));
        maxval = fmaxf(maxval, __shfl_down_sync(0xffffffff, maxval, 1, 32));
    }
    if (ty == 0)
    {
        reduction[0] = maxval;
    }
    __syncthreads();
    maxval = reduction[0];
    float divisor = 0.f;
#pragma unroll URF
    for (int i = ty; i<w/4; i+=BLOCK_DIM_Y)
    {
        float4 val = reinterpret_cast<float4*>(&a[row*w + i*4])[0];
        divisor += __expf(val.x - maxval);
        divisor += __expf(val.y - maxval);
        divisor += __expf(val.z - maxval);
        divisor += __expf(val.w - maxval);
    }

    divisor += __shfl_down_sync(0xffffffff, divisor, 16, 32);
    divisor += __shfl_down_sync(0xffffffff, divisor, 8, 32);
    divisor += __shfl_down_sync(0xffffffff, divisor, 4, 32);
    divisor += __shfl_down_sync(0xffffffff, divisor, 2, 32);
    divisor += __shfl_down_sync(0xffffffff, divisor, 1, 32);

    if (ty%32 == 0)
    {
      reduction[warp_id] = divisor;
    }

    __syncthreads();
    if (warp_id == 0)
    {
        divisor = ty < BLOCK_DIM_Y/32 ? reduction[ty] : 0;
        divisor += __shfl_down_sync(0xffffffff, divisor, 16, 32);
        divisor += __shfl_down_sync(0xffffffff, divisor, 8, 32);
        divisor += __shfl_down_sync(0xffffffff, divisor, 4, 32);
        divisor += __shfl_down_sync(0xffffffff, divisor, 2, 32);
        divisor += __shfl_down_sync(0xffffffff, divisor, 1, 32);
    }
    if (ty == 0)
    {
        reduction[0] = divisor;
    }

    __syncthreads();
    divisor = reduction[0];

#pragma unroll URF
    for (int i = ty; i<w/4; i+=BLOCK_DIM_Y)
    {
        float4 val = reinterpret_cast<float4*>(&a[row*w + i*4])[0];
        val.x = __expf(val.x-maxval)/divisor;
        val.y = __expf(val.y-maxval)/divisor;
        val.z = __expf(val.z-maxval)/divisor;
        val.w = __expf(val.w-maxval)/divisor;
        reinterpret_cast<float4*>(&b[row*w + i*4])[0] = val;
    }
  }
}

*/
template <typename scalar_t>
__global__ void softmax_kernel8(scalar_t* __restrict__ a, scalar_t* __restrict__ b, int w, int h)
{
  int row = blockIdx.x;
  int ty = threadIdx.y;
  int warp_id = ty/32;
  __shared__ float reduction_max[BLOCK_DIM_Y/32]; 
  __shared__ float reduction_div[BLOCK_DIM_Y/32]; 
  if (row < h)
  {
    float maxval = 0;
    float divisor = 0;
    float old_maxval = 0;
#pragma unroll URF
    for (int i = ty; i<w/4; i+=BLOCK_DIM_Y)
    {
        float4 val = reinterpret_cast<float4*>(&a[row*w + i*4])[0];
        maxval = fmaxf(maxval, val.x);
        maxval = fmaxf(maxval, val.y);
        maxval = fmaxf(maxval, val.z);
        maxval = fmaxf(maxval, val.w);
        if (maxval > old_maxval)
        {
          divisor *= __expf(old_maxval - maxval);
          old_maxval = maxval;
        }
        divisor += __expf(val.x - maxval);
        divisor += __expf(val.y - maxval);
        divisor += __expf(val.z - maxval);
        divisor += __expf(val.w - maxval);
    }
    float incoming_divisor = 0;
    float incoming_maxval = 0;
#pragma unroll URF
    for (int mask = 16; mask>0; mask/=2)
    {
      incoming_maxval = __shfl_xor_sync(0xffffffff, maxval, mask, 32);
      incoming_divisor = __shfl_xor_sync(0xffffffff, divisor, mask, 32);
      if (incoming_maxval > maxval)
      {
        divisor *= __expf(maxval - incoming_maxval);
        maxval = incoming_maxval;
      }
      else 
      {
        incoming_divisor *= __expf(incoming_maxval - maxval);
      }
      divisor += incoming_divisor;
    }

    if (ty%32 == 0)
    {
      reduction_max[warp_id] = maxval;
      reduction_div[warp_id] = divisor;
    }
    __syncthreads();
    if (warp_id == 0)
    {
        maxval = ty < BLOCK_DIM_Y/32 ? reduction_max[ty] : 0;
        divisor = ty < BLOCK_DIM_Y/32 ? reduction_div[ty] : 0;
#pragma unroll URF
        for (int mask = 16; mask>0; mask/=2)
        {
          incoming_maxval = __shfl_xor_sync(0xffffffff, maxval, mask, 32);
          incoming_divisor = __shfl_xor_sync(0xffffffff, divisor, mask, 32);
          if (incoming_maxval > maxval)
          {
            divisor *= __expf(maxval - incoming_maxval);
            maxval = incoming_maxval;
          }
          else 
          {
            incoming_divisor *= __expf(incoming_maxval - maxval);
          }
          divisor += incoming_divisor;
        }
    }
    if (ty == 0)
    {
        reduction_max[0] = maxval;
        reduction_div[0] = divisor;
    }
    __syncthreads();
    maxval = reduction_max[0];
    divisor = reduction_div[0];

#pragma unroll URF
    for (int i = ty; i<w/4; i+=BLOCK_DIM_Y)
    {
        float4 val = reinterpret_cast<float4*>(&a[row*w + i*4])[0];
        val.x = __expf(val.x-maxval)/divisor;
        val.y = __expf(val.y-maxval)/divisor;
        val.z = __expf(val.z-maxval)/divisor;
        val.w = __expf(val.w-maxval)/divisor;
        reinterpret_cast<float4*>(&b[row*w + i*4])[0] = val;
    }
  }
}
/*
template <typename scalar_t>
__global__ void softmax_kernel9(scalar_t* __restrict__ a, scalar_t* __restrict__ b, int w, int h)
{
  int row = blockIdx.x*blockDim.x + threadIdx.x;
  int ty = threadIdx.y;
  __shared__ float reduction[BLOCK_DIM_Y/2]; 
  if (row < h)
  {
    float maxval = 0;
    for (int i = ty; i<w/4; i+=BLOCK_DIM_Y)
    {
        float4 val = reinterpret_cast<float4*>(&a[row*w + i*4])[0];
        maxval = fmaxf(maxval, val.x);
        maxval = fmaxf(maxval, val.y);
        maxval = fmaxf(maxval, val.z);
        maxval = fmaxf(maxval, val.w);
    }

    if (ty >= BLOCK_DIM_Y/2)
    {
      reduction[ty - BLOCK_DIM_Y/2] = maxval;
    }
    #pragma unroll
    for(int stride = BLOCK_DIM_Y/2; stride>=1; stride/=2)
    {
      __syncthreads();
      if (ty < stride)
      {
        maxval = fmaxf(maxval, reduction[ty]);
        if (ty >= stride/2)
        {
          reduction[ty - stride/2] = maxval;
        }
      }
    }

    __syncthreads();
    maxval = reduction[0];

    float divisor = 0.f;
#pragma unroll URF
    for (int i = ty; i<w/4; i+=BLOCK_DIM_Y)
    {
        float4 val = reinterpret_cast<float4*>(&a[row*w + i*4])[0];
        divisor += __expf(val.x - maxval);
        divisor += __expf(val.y - maxval);
        divisor += __expf(val.z - maxval);
        divisor += __expf(val.w - maxval);
    }

    if (ty >= BLOCK_DIM_Y/2)
    {
      reduction[ty - BLOCK_DIM_Y/2] = divisor;
    }

#pragma unroll URF
    for(int stride = BLOCK_DIM_Y/2; stride>=1; stride/=2)
    {
      __syncthreads();
      if (ty < stride)
      {
        divisor = divisor + reduction[ty];
        if (ty >= stride/2)
        {
          reduction[ty - stride/2] = divisor;
        }
      }
    }
    __syncthreads();
    divisor = reduction[0];

#pragma unroll URF
    for (int i = ty; i<w/4; i+=BLOCK_DIM_Y)
    {
        float4 val = reinterpret_cast<float4*>(&a[row*w + i*4])[0];
        val.x = __expf(val.x-maxval)/divisor;
        val.y = __expf(val.y-maxval)/divisor;
        val.z = __expf(val.z-maxval)/divisor;
        val.w = __expf(val.w-maxval)/divisor;
        reinterpret_cast<float4*>(&b[row*w + i*4])[0] = val;
    }
  }
}


template <typename scalar_t>
__global__ void softmax_kernel10(scalar_t* __restrict__ a, scalar_t* __restrict__ b, int w, int h)
{
  int row = blockIdx.x;
  int ty = threadIdx.y;
  int warp_id = ty/32;
  int lane_id = ty%32;
  
  __shared__ float reduction[BLOCK_DIM_Y/32]; 
  float4 reg_array[CEILING((WIDTH/4),BLOCK_DIM_Y)];

  int reg_array_idx = 0;
  if (row < h)
  {
    float maxval = 0;
#pragma unroll URF
    for (int i = ty; i<WIDTH/4; i+=BLOCK_DIM_Y)
    {
      float4 val = reinterpret_cast<float4*>(&a[row*WIDTH + i*4])[0];
      maxval = fmaxf(maxval, val.x);
      maxval = fmaxf(maxval, val.y);
      maxval = fmaxf(maxval, val.z);
      maxval = fmaxf(maxval, val.w);
      reg_array[reg_array_idx] = val;
      reg_array_idx+=1;
    }
    maxval = fmaxf(maxval, __shfl_xor_sync(0xffffffff, maxval, 16, 32));
    maxval = fmaxf(maxval, __shfl_xor_sync(0xffffffff, maxval, 8, 32));
    maxval = fmaxf(maxval, __shfl_xor_sync(0xffffffff, maxval, 4, 32));
    maxval = fmaxf(maxval, __shfl_xor_sync(0xffffffff, maxval, 2, 32));
    maxval = fmaxf(maxval, __shfl_xor_sync(0xffffffff, maxval, 1, 32));

    if (lane_id == 0)
    {
      reduction[warp_id] = maxval;
    }
    __syncthreads();
    if (warp_id == 0)
    {
        maxval = ty < BLOCK_DIM_Y/32 ? reduction[ty] : 0;
        maxval = fmaxf(maxval, __shfl_xor_sync(0xffffffff, maxval, 16, 32));
        maxval = fmaxf(maxval, __shfl_xor_sync(0xffffffff, maxval, 8, 32));
        maxval = fmaxf(maxval, __shfl_xor_sync(0xffffffff, maxval, 4, 32));
        maxval = fmaxf(maxval, __shfl_xor_sync(0xffffffff, maxval, 2, 32));
        maxval = fmaxf(maxval, __shfl_xor_sync(0xffffffff, maxval, 1, 32));
    }
    if (ty == 0)
    {
        reduction[0] = maxval;
    }
    __syncthreads();
    maxval = reduction[0];
    float divisor = 0.f;
    reg_array_idx=0;
#pragma unroll URF
    for (int i = ty; i<WIDTH/4; i+=BLOCK_DIM_Y)
    {
        float4 val = reg_array[reg_array_idx];
        val.x = __expf(val.x - maxval);
        val.y = __expf(val.y - maxval);
        val.z = __expf(val.z - maxval);
        val.w = __expf(val.w - maxval);
        divisor += val.x;
        divisor += val.y;
        divisor += val.z;
        divisor += val.w;
        reg_array[reg_array_idx] = val;
      reg_array_idx+=1;
    }

    divisor += __shfl_xor_sync(0xffffffff, divisor, 16, 32);
    divisor += __shfl_xor_sync(0xffffffff, divisor, 8, 32);
    divisor += __shfl_xor_sync(0xffffffff, divisor, 4, 32);
    divisor += __shfl_xor_sync(0xffffffff, divisor, 2, 32);
    divisor += __shfl_xor_sync(0xffffffff, divisor, 1, 32);

    if (lane_id == 0)
    {
      reduction[warp_id] = divisor;
    }

    __syncthreads();
    if (warp_id == 0)
    {
        divisor = ty < BLOCK_DIM_Y/32 ? reduction[ty] : 0;
        divisor += __shfl_xor_sync(0xffffffff, divisor, 16, 32);
        divisor += __shfl_xor_sync(0xffffffff, divisor, 8, 32);
        divisor += __shfl_xor_sync(0xffffffff, divisor, 4);
        divisor += __shfl_xor_sync(0xffffffff, divisor, 2);
        divisor += __shfl_xor_sync(0xffffffff, divisor, 1);
    }

    if (ty == 0)
    {
        reduction[0] = divisor;
    }

    __syncthreads();
    divisor = reduction[0];

    reg_array_idx = 0;
#pragma unroll URF
    for (int i = ty; i<WIDTH/4; i+=BLOCK_DIM_Y)
    {
        float4 val = reg_array[reg_array_idx];
        val.x = val.x/divisor;
        val.y = val.y/divisor;
        val.z = val.z/divisor;
        val.w = val.w/divisor;
        reinterpret_cast<float4*>(&b[row*WIDTH + i*4])[0] = val;
      reg_array_idx+=1;
    }

  }
}
*/
torch::Tensor softmax_cu(torch::Tensor x)
{
  auto out = torch::empty_like(x);
  int h = x.size(0);
  int w = x.size(1);

  dim3 block_size = dim3(1, BLOCK_DIM_Y, 1);
  dim3 grid_size = dim3(h, 1, 1);
/*
#if SOFTMAX_VARIANT == 1
  block_size = dim3(32, 32, 1);
  grid_size = dim3(w/32, h/32, 1);
  AT_DISPATCH_FLOATING_TYPES(x.type(), "softmax_cuda", ([&] {
        softmax_kernel<scalar_t><<<grid_size, block_size>>>
          (x.data_ptr<scalar_t>(), out.data_ptr<scalar_t>(), w, h);
        }));
#endif
#if SOFTMAX_VARIANT == 2
  AT_DISPATCH_FLOATING_TYPES(x.type(), "softmax_cuda", ([&] {
        softmax_kernel2<scalar_t><<<grid_size, block_size>>>
          (x.data_ptr<scalar_t>(), out.data_ptr<scalar_t>(), w, h);
        }));
#endif
#if SOFTMAX_VARIANT == 3
  AT_DISPATCH_FLOATING_TYPES(x.type(), "softmax_cuda", ([&] {
        softmax_kernel3<scalar_t><<<grid_size, block_size>>>
          (x.data_ptr<scalar_t>(), out.data_ptr<scalar_t>(), w, h);
        }));
#endif*/
#if SOFTMAX_VARIANT == 4
  AT_DISPATCH_FLOATING_TYPES(x.type(), "softmax_cuda", ([&] {
        softmax_kernel4<scalar_t><<<grid_size, block_size>>>
          (x.data_ptr<scalar_t>(), out.data_ptr<scalar_t>(), w, h);
        }));
#endif/*
#if SOFTMAX_VARIANT == 5
  AT_DISPATCH_FLOATING_TYPES(x.type(), "softmax_cuda", ([&] {
        softmax_kernel5<scalar_t><<<grid_size, block_size>>>
          (x.data_ptr<scalar_t>(), out.data_ptr<scalar_t>(), w, h);
        }));
#endif
#if SOFTMAX_VARIANT == 6
  AT_DISPATCH_FLOATING_TYPES(x.type(), "softmax_cuda", ([&] {
        softmax_kernel6<scalar_t><<<grid_size, block_size>>>
          (x.data_ptr<scalar_t>(), out.data_ptr<scalar_t>(), w, h);
        }));
#endif
#if SOFTMAX_VARIANT == 7
  AT_DISPATCH_FLOATING_TYPES(x.type(), "softmax_cuda", ([&] {
        softmax_kernel7<scalar_t><<<grid_size, block_size>>>
          (x.data_ptr<scalar_t>(), out.data_ptr<scalar_t>(), w, h);
        }));
#endif*/
#if SOFTMAX_VARIANT == 8
  AT_DISPATCH_FLOATING_TYPES(x.type(), "softmax_cuda", ([&] {
        softmax_kernel8<scalar_t><<<grid_size, block_size>>>
          (x.data_ptr<scalar_t>(), out.data_ptr<scalar_t>(), w, h);
        }));
#endif
/*
#if SOFTMAX_VARIANT == 9
  AT_DISPATCH_FLOATING_TYPES(x.type(), "softmax_cuda", ([&] {
        softmax_kernel9<scalar_t><<<grid_size, block_size>>>
          (x.data_ptr<scalar_t>(), out.data_ptr<scalar_t>(), w, h);
        }));
#endif

#if SOFTMAX_VARIANT == 10
  AT_DISPATCH_FLOATING_TYPES(x.type(), "softmax_cuda", ([&] {
        softmax_kernel10<scalar_t><<<grid_size, block_size>>>
          (x.data_ptr<scalar_t>(), out.data_ptr<scalar_t>(), w, h);
        }));
#endif
*/
  return out;
}
